#include "hip/hip_runtime.h"
#define BLOCK_DIM 1024

__global__ void scan_kernel(float* input, float* output, float* partialSums, unsigned int N) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float buffer1_s[BLOCK_DIM];
    __shared__ float buffer2_s[BLOCK_DIM];
    float* inBuffer_s = buffer1_s;
    float* outBuffer_s = buffer2_s;

    if (i < N) {
        inBuffer_s[threadIdx.x] = input[i];
    } else {
        inBuffer_s[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM / 2; stride *= 2) {
        if (threadIdx.x >= stride) {
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x] + inBuffer_s[threadIdx.x - stride];
        } else {
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x];
        }
        __syncthreads();
        float* tmp = inBuffer_s;
        inBuffer_s = outBuffer_s;
        outBuffer_s = tmp;
    }

    if (threadIdx.x == BLOCK_DIM - 1) {
        partialSums[blockIdx.x] = buffer_s[threadIdx.x];
    }
    output[i] = inBuffer_s[threadIdx.x];
}
