#include "hip/hip_runtime.h"
#define BLOCK_DIM 1024

__global__ void scan_kernel(float* input, float* output, float* partialSums, unsigned int N) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float buffer1_s[BLOCK_DIM];
    __shared__ float buffer2_s[BLOCK_DIM];
    float* inBuffer_s = buffer1_s;
    float* outBuffer_s = buffer2_s;
    inbuffer_s[threadIdx.x] = input[i];
    __syncthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM / 2; stride *= 2) {
        if (threadIdx.x >= stride) {
            outbuffer_s[threadIdx.x] = inbuffer_s[threadIdx.x] + inbuffer_s[threadIdx.x - stride];
        } 
        else {
            outbuffer_s[threadIdx.x] = inbuffer_s[threadIdx.x];
        }
        __syncthreads();
        float* tmp = inBuffer_s;
        inBuffer_s = outBuffer_s;
        outBuffer_s = tmp;
    }

    if (threadIdx.x == BLOCK_DIM - 1) {
        partialSums[blockIdx.x] = buffer_s[threadIdx.x];
    }
    output[i] = inbuffer_s[threadIdx.x];
}
